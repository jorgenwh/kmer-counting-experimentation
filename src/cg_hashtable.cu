#include <iostream>
#include <sstream>
#include <inttypes.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "cg_hashtable.h"

CGHashTable::CGHashTable(const uint64_t *keys, const bool cuda_keys, const uint32_t size, const uint32_t capacity) {
  initialize(keys, cuda_keys, size, capacity);
}

void CGHashTable::initialize(const uint64_t *keys, const bool cuda_keys, const uint32_t size, const uint32_t capacity) {
  capacity_m = capacity;
  size_m = size;

  // Allocate the table
  cuda_errchk(hipMalloc(&table_m.keys, sizeof(uint64_t)*capacity));
  cuda_errchk(hipMemset(table_m.keys, 0xff, sizeof(uint64_t)*capacity));
  cuda_errchk(hipMalloc(&table_m.values, sizeof(uint32_t)*capacity));
  cuda_errchk(hipMemset(table_m.values, 0xff, sizeof(uint32_t)*capacity));

  uint64_t *d_keys;
  if (!cuda_keys) {
    cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
    cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  init_hashtable(table_m, cuda_keys ? keys : d_keys, size, capacity);

  if (!cuda_keys) {
    cuda_errchk(hipFree(d_keys));
  }
}

void CGHashTable::get(const uint64_t *keys, uint32_t *counts, uint32_t size) const {
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  uint32_t *d_counts;
  cuda_errchk(hipMalloc(&d_counts, sizeof(uint32_t)*size));

  lookup_hashtable(table_m, d_keys, d_counts, size, capacity_m); 

  cuda_errchk(hipMemcpy(counts, d_counts, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));
  cuda_errchk(hipFree(d_keys));
  cuda_errchk(hipFree(d_counts));
}

void CGHashTable::getcu(const uint64_t *keys, uint32_t *counts, uint32_t size) const {
  lookup_hashtable(table_m, keys, counts, size, capacity_m); 
}

void CGHashTable::count(const uint64_t *keys, const uint32_t size) {
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  count_hashtable(table_m, d_keys, size, capacity_m);
  cuda_errchk(hipFree(d_keys));
}

void CGHashTable::countcu(const uint64_t *keys, const uint32_t size) {
  count_hashtable(table_m, keys, size, capacity_m);
}

std::string CGHashTable::to_string() const {
  int print_size = (capacity_m < 40) ? capacity_m : 40;

  uint64_t *keys = new uint64_t[capacity_m];
  uint32_t *values = new uint32_t[capacity_m];
  cuda_errchk(hipMemcpy(keys, table_m.keys, sizeof(uint64_t)*capacity_m, hipMemcpyDeviceToHost));
  cuda_errchk(hipMemcpy(values, table_m.values, sizeof(uint32_t)*capacity_m, hipMemcpyDeviceToHost));

  std::ostringstream oss;
  std::ostringstream keys_oss;
  std::ostringstream values_oss;

  keys_oss << "[";
  values_oss << "[";
  uint32_t elements = 0;
  for (int i = 0; i < capacity_m; i++) {
    uint64_t key = keys[i];
    uint32_t value = values[i];

    if (key == kEmpty) { continue; }

    if (elements != 0) { 
      keys_oss << ", "; 
      values_oss << ", "; 
    }

    keys_oss << key;
    values_oss << value;
    
    elements++;
    if (elements >= print_size) { break; }
  }
  keys_oss << "]";
  values_oss << "]";

  oss << "Counter(" << keys_oss.str() << ", " << values_oss.str();
  oss << ", size=" << size_m << ", capacity=" << capacity_m << ")";

  delete[] keys;
  delete[] values;

  return oss.str();
}
