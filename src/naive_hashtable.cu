#include <iostream>
#include <sstream>
#include <inttypes.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "naive_hashtable.h"

NaiveHashTable::NaiveHashTable(const uint64_t *keys, const uint32_t size, const uint32_t capacity) {
  initialize(keys, size, capacity);
}

void NaiveHashTable::initialize(const uint64_t *keys, const uint32_t size, const uint32_t capacity) {
  capacity_m = capacity;
  size_m = size;

  cuda_errchk(hipMalloc(&table_m, sizeof(KeyValue)*capacity));
  cuda_errchk(hipMemset(table_m, 0xff, sizeof(KeyValue)*capacity));
  cuda_errchk(hipDeviceSynchronize());

  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  init_hashtable(table_m, d_keys, size, capacity);
  cuda_errchk(hipFree(d_keys));
}

void NaiveHashTable::get(const uint64_t *keys, uint64_t *counts, uint32_t size) const {
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  uint64_t *d_counts;
  cuda_errchk(hipMalloc(&d_counts, sizeof(uint64_t)*size));

  lookup_hashtable(table_m, d_keys, d_counts, size, capacity_m); 

  cuda_errchk(hipMemcpy(counts, d_counts, sizeof(uint64_t)*size, hipMemcpyDeviceToHost));
  cuda_errchk(hipFree(d_keys));
  cuda_errchk(hipFree(d_counts));
}

void NaiveHashTable::getcu(const uint64_t *keys, uint64_t *counts, uint32_t size) const {
  lookup_hashtable(table_m, keys, counts, size, capacity_m); 
}

void NaiveHashTable::count(const uint64_t *keys, const uint32_t size) {
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  count_hashtable(table_m, d_keys, size, capacity_m);
  cuda_errchk(hipFree(d_keys));
}

void NaiveHashTable::countcu(const uint64_t *keys, const uint32_t size) {
  count_hashtable(table_m, keys, size, capacity_m);
}

std::string NaiveHashTable::to_string() const {
  int print_size = (capacity_m < 100) ? capacity_m : 100;

  KeyValue *h_table = new KeyValue[capacity_m];
  cuda_errchk(hipMemcpy(h_table, table_m, sizeof(KeyValue)*capacity_m, hipMemcpyDeviceToHost));

  std::ostringstream oss;
  std::ostringstream keys_oss;
  std::ostringstream values_oss;

  keys_oss << "[";
  values_oss << "[";
  uint32_t elements = 0;
  for (int i = 0; i < capacity_m; i++) {
    KeyValue cur = h_table[i];
    if (cur.key == kEmpty) { continue; }

    if (elements != 0) { 
      keys_oss << ", "; 
      values_oss << ", "; 
    }

    keys_oss << cur.key;
    values_oss << cur.value;
    
    elements++;
    if (elements >= print_size) { break; }
  }
  keys_oss << "]";
  values_oss << "]";

  oss << "Counter(" << keys_oss.str() << ", " << values_oss.str();
  oss << ", capacity=" << capacity_m << ")";

  delete[] h_table;
  return oss.str();
}
