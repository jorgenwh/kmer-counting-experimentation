#include <iostream>
#include <sstream>
#include <inttypes.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "cops_kernels.h"
#include "cops_hashtable.h"

COPSHashTable::COPSHashTable(const uint64_t *keys, const bool cuda_keys, const uint32_t size, const uint32_t capacity, const uint32_t cg_size) {
  initialize(keys, cuda_keys, size, capacity, cg_size);
}

void COPSHashTable::initialize(const uint64_t *keys, const bool cuda_keys, const uint32_t size, const uint32_t capacity, const uint32_t cg_size) {
  capacity_m = capacity;
  size_m = size;
  cg_size_m = cg_size;

  // Allocate the table
  cuda_errchk(hipMalloc(&table_m.keys, sizeof(uint64_t)*capacity));
  cuda_errchk(hipMemset(table_m.keys, 0xff, sizeof(uint64_t)*capacity));
  cuda_errchk(hipMalloc(&table_m.values, sizeof(uint32_t)*capacity));
  cuda_errchk(hipMemset(table_m.values, 0xff, sizeof(uint32_t)*capacity));

  uint64_t *d_keys;
  if (!cuda_keys) {
    cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
    cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  cops_kernels::init_hashtable(table_m, cuda_keys ? keys : d_keys, size, capacity, cg_size_m);

  if (!cuda_keys) {
    cuda_errchk(hipFree(d_keys));
  }
}

void COPSHashTable::get(const uint64_t *keys, uint32_t *counts, uint32_t size) const {
  // Copy the keys being looked up to the device
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  // Allocate memory to store the values being looked up
  uint32_t *d_counts;
  cuda_errchk(hipMalloc(&d_counts, sizeof(uint32_t)*size));

  cops_kernels::lookup_hashtable(table_m, d_keys, d_counts, size, capacity_m, cg_size_m); 

  // Copy the looked up values back to host and free keys and values from device memory
  cuda_errchk(hipMemcpy(counts, d_counts, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));
  cuda_errchk(hipFree(d_keys));
  cuda_errchk(hipFree(d_counts));
}

void COPSHashTable::getcu(const uint64_t *keys, uint32_t *counts, uint32_t size) const {
  cops_kernels::lookup_hashtable(table_m, keys, counts, size, capacity_m, cg_size_m); 
}

void COPSHashTable::count(const uint64_t *keys, const uint32_t size) {
  // Copy the keys being counted to the device
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  cops_kernels::count_hashtable(table_m, d_keys, size, capacity_m, cg_size_m);
  cuda_errchk(hipFree(d_keys));
}

void COPSHashTable::countcu(const uint64_t *keys, const uint32_t size) {
  cops_kernels::count_hashtable(table_m, keys, size, capacity_m, cg_size_m);
}

std::string COPSHashTable::to_string() const {
  int print_size = (capacity_m < 40) ? capacity_m : 40;

  uint64_t *keys = new uint64_t[capacity_m];
  uint32_t *values = new uint32_t[capacity_m];
  cuda_errchk(hipMemcpy(keys, table_m.keys, sizeof(uint64_t)*capacity_m, hipMemcpyDeviceToHost));
  cuda_errchk(hipMemcpy(values, table_m.values, sizeof(uint32_t)*capacity_m, hipMemcpyDeviceToHost));

  std::ostringstream oss;
  std::ostringstream keys_oss;
  std::ostringstream values_oss;

  keys_oss << "[";
  values_oss << "[";
  uint32_t elements = 0;
  for (int i = 0; i < capacity_m; i++) {
    uint64_t key = keys[i];
    uint32_t value = values[i];

    if (key == kEmpty) { continue; }

    if (elements != 0) { 
      keys_oss << ", "; 
      values_oss << ", "; 
    }

    keys_oss << key;
    values_oss << value;
    
    elements++;
    if (elements >= print_size) { break; }
  }
  keys_oss << "]";
  values_oss << "]";

  oss << "CopsHashTable(" << keys_oss.str() << ", " << values_oss.str();
  oss << ", size=" << size_m << ", capacity=" << capacity_m << ")";

  delete[] keys;
  delete[] values;

  return oss.str();
}
