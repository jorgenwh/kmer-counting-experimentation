#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "cu_common.h"
#include "kernels.h"

__device__ inline uint64_t word_reverse_complement(const uint64_t kmer, uint8_t kmer_size) {
  uint64_t res = ~kmer;
  res = ((res >> 2 & 0x3333333333333333) | (res & 0x3333333333333333) << 2);
  res = ((res >> 4 & 0x0F0F0F0F0F0F0F0F) | (res & 0x0F0F0F0F0F0F0F0F) << 4);
  res = ((res >> 8 & 0x00FF00FF00FF00FF) | (res & 0x00FF00FF00FF00FF) << 8);
  res = ((res >> 16 & 0x0000FFFF0000FFFF) | (res & 0x0000FFFF0000FFFF) << 16);
  res = ((res >> 32 & 0x00000000FFFFFFFF) | (res & 0x00000000FFFFFFFF) << 32);
  return (res >> (2 * (32 - kmer_size)));
}

__global__ void init_hashtable_kernel(
    Table table, const uint64_t *keys, const uint32_t size, const uint32_t capacity) {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id < size) {
    uint64_t key = keys[thread_id];
    uint64_t hash = key % capacity;

    while (true) {
      unsigned long long int *old_ptr = reinterpret_cast<unsigned long long int *>(&table.keys[hash]);
      uint64_t old = atomicCAS(old_ptr, kEmpty, key);

      if (old == kEmpty || old == key) {
        table.values[hash] = 0;
        return;
      }
      hash = (hash + 1) % capacity;
    }
  }
}

void init_hashtable(
    Table table, const uint64_t *keys, const uint32_t size, const uint32_t capacity) {
  int min_grid_size;
  int thread_block_size = 512;
  /*cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      init_hashtable_kernel, 0, 0));*/

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  init_hashtable_kernel<<<grid_size, thread_block_size>>>(table, keys, size, capacity);
  //cuda_errchk(hipDeviceSynchronize());
}

__global__ void lookup_hashtable_kernel(Table table, 
    const uint64_t *keys, uint32_t *counts, const uint32_t size, const uint32_t capacity) {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id < size) {
    uint64_t key = keys[thread_id];
    uint64_t hash = key % capacity;

    while (true) {
      uint64_t cur_key = table.keys[hash];
      if (cur_key == key || cur_key == kEmpty) {
        counts[thread_id] = (cur_key == key) ? table.values[hash] : 0;
        return;
      }
      hash = (hash + 1) % capacity;
    }
  }
}

void lookup_hashtable(Table table, 
    const uint64_t *keys, uint32_t *counts, const uint32_t size, const uint32_t capacity) {
  int min_grid_size;
  int thread_block_size = 512;
  /*cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      lookup_hashtable_kernel, 0, 0));*/

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  lookup_hashtable_kernel<<<grid_size, thread_block_size>>>(table, keys, counts, size, capacity);
  //cuda_errchk(hipDeviceSynchronize());
}

__global__ void count_hashtable_kernel(
    Table table, const uint64_t *keys, const uint32_t size, const uint32_t capacity) {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id < size) {
    uint64_t key = keys[thread_id];
    uint64_t hash = key % capacity;

    //printf("%llu -> %llu\n", key, revcomp);

    // Search for original key
    while (true) {
      uint64_t cur_key = table.keys[hash];
      if (cur_key == kEmpty) { break; }
      if (cur_key == key) {
        atomicAdd((unsigned int *)&(table.values[hash]), 1);
        return;
      }

      hash = (hash + 1) % capacity;
    }

    key = word_reverse_complement(key, 31);
    hash = key % capacity;

    // Search for reverse complement of key
    while (true) {
      uint64_t cur_key = table.keys[hash];
      if (cur_key == kEmpty) { return; }
      if (cur_key == key) {
        atomicAdd((unsigned int *)&(table.values[hash]), 1);
        return;
      }

      hash = (hash + 1) % capacity;
    }
  }
}

void count_hashtable(
    Table table, const uint64_t *keys, const uint32_t size, const uint32_t capacity) {
  int min_grid_size;
  int thread_block_size = 512;
  /*cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      count_hashtable_kernel, 0, 0));*/

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  count_hashtable_kernel<<<grid_size, thread_block_size>>>(table, keys, size, capacity);
  //cuda_errchk(hipDeviceSynchronize());
}

